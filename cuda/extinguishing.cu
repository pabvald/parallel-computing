#include "hip/hip_runtime.h"
/*
 * Simplified simulation of fire extinguishing
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2018/2019
 *
 * v1.4
 *
 * (c) 2019 Arturo Gonzalez Escribano
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include "cputils.h"
#include <hip/hip_runtime.h>

#define RADIUS_TYPE_1		3
#define RADIUS_TYPE_2_3		9
#define THRESHOLD	0.1f

/* Structure to store data of an extinguishing team */
typedef struct {
	int x,y;
	int type;
	int target;
} Team;

/* Structure to store data of a fire focal point */
typedef struct {
	int x,y;
	int start;
	int heat;
	int active; // States: 0 Not yet activated; 1 Active; 2 Deactivated by a team
} FocalPoint;

/* Macro function to simplify accessing with two coordinates to a flattened array */
#define accessMat( arr, exp1, exp2 )	arr[ (exp1) * paddingColumns + (exp2) ]

/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s <config_file> | <command_line_args>\n", program_name );
	fprintf(stderr,"\t<config_file> ::= -f <file_name>\n");
	fprintf(stderr,"\t<command_line_args> ::= <rows> <columns> <maxIter> <numTeams> [ <teamX> <teamY> <teamType> ... ] <numFocalPoints> [ <focalX> <focalY> <focalStart> <focalTemperature> ... ]\n");
	fprintf(stderr,"\n");
}

#ifdef DEBUG
/* 
 * Function: Print the current state of the simulation 
 */
void print_status( int iteration, int rows, int columns, float *surface, int num_teams, Team *teams, int num_focal, FocalPoint *focal, float global_residual ) {
	/* 
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( surface, i, j ) >= 1000 ) symbol = '*';
			else if ( accessMat( surface, i, j ) >= 100 ) symbol = '0' + (int)(accessMat( surface, i, j )/100);
			else if ( accessMat( surface, i, j ) >= 50 ) symbol = '+';
			else if ( accessMat( surface, i, j ) >= 25 ) symbol = '.';
			else symbol = '0';

			int t;
			int flag_team = 0;
			for( t=0; t<num_teams; t++ ) 
				if ( teams[t].x == i && teams[t].y == j ) { flag_team = 1; break; }
			if ( flag_team ) printf("[%c]", symbol );
			else {
				int f;
				int flag_focal = 0;
				for( f=0; f<num_focal; f++ ) 
					if ( focal[f].x == i && focal[f].y == j && focal[f].active == 1 ) { flag_focal = 1; break; }
				if ( flag_focal ) printf("(%c)", symbol );
				else printf(" %c ", symbol );
			}
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Global residual: %f\n\n", global_residual);
}
#endif



/**************************************************************** ERROR CHECKING **************************************************************************/

/**
 * Check if an error has ocurred, in which case print the error
 * and a given message, and exit.
 * @param err - cuda error
 * @param s - message (indicating where the error ocurred)
 */
 void CHECK_ERROR(const char* s) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("--ERROR (%s): %s\n", s, hipGetErrorString(err));
		exit( EXIT_FAILURE );
	}
}



/******************************************************************* KERNELS *****************************************************************************/

/**
 * Kernel 1
 * Update the surface(skip borders)
 *
 */
__global__ void propagate_kernel(float *surface, float *surfaceCopy, int rows, int columns, int paddingColumns) {

	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);

	if (gid >= rows*paddingColumns) return;

	int i = gid / paddingColumns;
	int j = gid % paddingColumns;

	if (i <= 0 || i >= rows-1 || j <= 0 || j >= columns-1) return; // Out of heated surface 

	accessMat( surface, i, j ) = ( 
									accessMat( surfaceCopy, i-1, j ) +
									accessMat( surfaceCopy, i+1, j ) +
									accessMat( surfaceCopy, i, j-1 ) +
									accessMat( surfaceCopy, i, j+1 ) ) / 4;
}

/**
 * Kernel 2
 * Compute the residual difference (absolute value).
 */
__global__ void difference_kernel(float *global_residual, float *surface, float * surfaceCopy, int rows, int columns, int paddingColumns) {

	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);

	if (gid >= rows*paddingColumns) return;

	int i = gid / paddingColumns;
	int j = gid % paddingColumns;

	if (i <= 0 || i >= rows-1 || j <= 0 || j >= columns-1) {
		accessMat( global_residual, i, j ) = 0;
		return; // Out of heated surface 
	} 
	accessMat( global_residual, i, j ) = fabs(accessMat( surface, i, j) - accessMat( surfaceCopy, i, j ));
}

/**
 * Kernel 3
 * Get the maximum value in data
 */ 
__global__ void reduce_max1_kernel(float *data, int size) {

	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + (threadIdx.y * blockDim.x + threadIdx.x);
    
    if(gid >= size/2)     return;

	if (data[gid] < data[gid + size/2]) {
        data[gid] = data[gid + size/2];
    }

    /*In case the reduction size is odd, there will be a mismatched element. The last thread
      will have to cover it as well */
    if (size % 2 != 0) {
        if (gid == size/2 - 1) {
            if (data[gid] < data[size - 1]) {
                data[gid] = data[size - 1];
            }
        }
    }
}

/**
 * Kernel 4
 * Get the maximum value in data.
 */
__global__ void reduce_max2_kernel(float* data, int size) {
    
    // Shared memory 
    extern __shared__ float tmp[];
    
    int gid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
    if (gid >= size) return; 

    // Load data in shared memory 
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    tmp[tid] = data[gid];
    
    __syncthreads();

	int mysize = blockDim.x;
	
	if ( (blockIdx.x == gridDim.x-1) && ( (blockDim.x * gridDim.x - size) > 0))
	{
	 	mysize = blockDim.x - (blockDim.x * gridDim.x - size);
	}

    // Reduction in shared memory 
    for (unsigned int s=mysize/2; s>0; s/=2)
	{   
		if (tid < s ) {  
            if (tmp[tid+s] > tmp[tid]) {
				tmp[tid] = tmp[tid+s];  // max(tmp[tid], tmp[tid+s])
			}  			
		}        
		
		// If size is not even 
		if ( (size % 2 != 0) && tid == 0 && tmp[size-1] > tmp[0]  ) {
			tmp[0] = tmp[size-1]; 
		}

		__syncthreads();
    }
    
    /* The thread 0 of each block writes the final result of the reduction
     * in the device's global memory given as a parameter (g_odata[]) */
    if (tid == 0) {
        data[blockIdx.x] = tmp[tid];
    }
}


/**
 * Kernel 5
 * Reduce the heat around the team. 
 */
__global__ void team_actions_kernel(float *surface, int rows, int columns, int paddingColumns, int teamX, int teamY, int squared_radius) {

	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
		
	if (gid >= rows*paddingColumns) return;

	int i = gid / paddingColumns;
	int j = gid % paddingColumns;

	if (i <= 0 || i >= rows-1 || j <= 0 || j >= columns-1) return; // Out of heated surface 

	if ( (teamX - i)*(teamX - i) + (teamY-j)*(teamY-j) >squared_radius) return ;

	accessMat( surface, i, j) = accessMat( surface, i, j ) * 0.75;
}


/**
 * Kernel 6
 * Update the heat of the surface.
 */
__global__ void update_heat_kernel(float *surface, int rows, int paddingColumns, int x, int y, int heat) {

	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);
		
	if (gid >= rows*paddingColumns) return;

	int i = gid / paddingColumns;
	int j = gid % paddingColumns;

	if (i == x || j == y) {
		accessMat(surface, i, j) = heat;
	}
}

 /*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	int i,j,t;

	// Simulation data
	int rows, columns, max_iter;
	float *surface, *surfaceCopy;
	int num_teams, num_focal;
	Team *teams;
	FocalPoint *focal;

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc<2) {
		fprintf(stderr,"-- Error in arguments: No arguments\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	int read_from_file = ! strcmp( argv[1], "-f" );
	/* 1.2. Read configuration from file */
	if ( read_from_file ) {
		/* 1.2.1. Open file */
		if (argc<3) {
			fprintf(stderr,"-- Error in arguments: file-name argument missing\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		FILE *args = cp_abrir_fichero( argv[2] );
		if ( args == NULL ) {
			fprintf(stderr,"-- Error in file: not found: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}	

		/* 1.2.2. Read surface and maximum number of iterations */
		int ok;
		ok = fscanf(args, "%d %d %d", &rows, &columns, &max_iter);
		if ( ok != 3 ) {
			fprintf(stderr,"-- Error in file: reading rows, columns, max_iter from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		if ( surface == NULL || surfaceCopy == NULL ) {
			fprintf(stderr,"-- Error allocating: surface structures\n");
			exit( EXIT_FAILURE );
		}

		/* 1.2.3. Teams information */
		ok = fscanf(args, "%d", &num_teams );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error file, reading num_teams from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			ok = fscanf(args, "%d %d %d", &teams[i].x, &teams[i].y, &teams[i].type);
			if ( ok != 3 ) {
				fprintf(stderr,"-- Error in file: reading team %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
		}

		/* 1.2.4. Focal points information */
		ok = fscanf(args, "%d", &num_focal );
		if ( ok != 1 ) {
			fprintf(stderr,"-- Error in file: reading num_focal from file: %s\n", argv[1]);
			exit( EXIT_FAILURE );
		}
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( focal == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			ok = fscanf(args, "%d %d %d %d", &focal[i].x, &focal[i].y, &focal[i].start, &focal[i].heat);
			if ( ok != 4 ) {
				fprintf(stderr,"-- Error in file: reading focal point %d from file: %s\n", i, argv[1]);
				exit( EXIT_FAILURE );
			}
			focal[i].active = 0;
		}
	}
	/* 1.3. Read configuration from arguments */
	else {
		/* 1.3.1. Check minimum number of arguments */
		if (argc<6) {
			fprintf(stderr, "-- Error in arguments: not enough arguments when reading configuration from the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}

		/* 1.3.2. Surface and maximum number of iterations */
		rows = atoi( argv[1] );
		columns = atoi( argv[2] );
		max_iter = atoi( argv[3] );

		surface = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );
		surfaceCopy = (float *)malloc( sizeof(float) * (size_t)rows * (size_t)columns );

		/* 1.3.3. Teams information */
		num_teams = atoi( argv[4] );
		teams = (Team *)malloc( sizeof(Team) * (size_t)num_teams );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		if ( argc < num_teams*3 + 5 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d teams\n", num_teams );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_teams; i++ ) {
			teams[i].x = atoi( argv[5+i*3] );
			teams[i].y = atoi( argv[6+i*3] );
			teams[i].type = atoi( argv[7+i*3] );
		}

		/* 1.3.4. Focal points information */
		int focal_args = 5 + i*3;
		if ( argc < focal_args+1 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for the number of focal points\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		num_focal = atoi( argv[focal_args] );
		focal = (FocalPoint *)malloc( sizeof(FocalPoint) * (size_t)num_focal );
		if ( teams == NULL ) {
			fprintf(stderr,"-- Error allocating: %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		if ( argc < focal_args + 1 + num_focal*4 ) {
			fprintf(stderr,"-- Error in arguments: not enough arguments for %d focal points\n", num_focal );
			exit( EXIT_FAILURE );
		}
		for( i=0; i<num_focal; i++ ) {
			focal[i].x = atoi( argv[focal_args+i*4+1] );
			focal[i].y = atoi( argv[focal_args+i*4+2] );
			focal[i].start = atoi( argv[focal_args+i*4+3] );
			focal[i].heat = atoi( argv[focal_args+i*4+4] );
			focal[i].active = 0;
		}

		/* 1.3.5. Sanity check: No extra arguments at the end of line */
		if ( argc > focal_args+i*4+1 ) {
			fprintf(stderr,"-- Error in arguments: extra arguments at the end of the command line\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Teams: %d, Focal points: %d\n", num_teams, num_focal );
	for( i=0; i<num_teams; i++ ) {
		printf("\tTeam %d, position (%d,%d), type: %d\n", i, teams[i].x, teams[i].y, teams[i].type );
	}
	for( i=0; i<num_focal; i++ ) {
		printf("\tFocal_point %d, position (%d,%d), start time: %d, temperature: %d\n", i, 
		focal[i].x,
		focal[i].y,
		focal[i].start,
		focal[i].heat );
	}
#endif // DEBUG

	/* 2. Select GPU and start global timer */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */
	/* GLOBAL variables */
	int SQUARED_RADIUS_TYPE_1 = RADIUS_TYPE_1 * RADIUS_TYPE_1;
	int SQUARED_RADIUS_TYPE_2_3 = RADIUS_TYPE_2_3 * RADIUS_TYPE_2_3;

	int TRANSACTION_SEGMENT_BYTES = 128;
	int NUM_THREADS_PER_BLOCK = 128;


	/* Make columns a multiple of TRANSACTION_SEGMENT_BYTES */
	int paddingColumns;
	if (columns % (TRANSACTION_SEGMENT_BYTES/sizeof(float)) == 0) {
		paddingColumns = columns;

	} else {
		paddingColumns = (columns / (TRANSACTION_SEGMENT_BYTES/sizeof(float)) + 1 ) * (TRANSACTION_SEGMENT_BYTES/sizeof(float));
		
		surface = (float *) realloc(surface, rows * paddingColumns * sizeof(float));
		surfaceCopy = (float *) realloc(surfaceCopy, rows * paddingColumns * sizeof(float));

		if ( surface == NULL || surfaceCopy == NULL ) {
			fprintf(stderr,"-- Error RE-allocating: surface structures\n");
			exit( EXIT_FAILURE );
		}
	}
	
	

	/* Geometry of grids and blocks */
	int realNumValues = rows * paddingColumns;
	//int nearestUpperPow2 = pow(2,ceil(log2((double) realNumValues)));
	int numValues = realNumValues;

	int numBlocks = numValues / NUM_THREADS_PER_BLOCK;
	if (numValues % NUM_THREADS_PER_BLOCK != 0) {
		numBlocks++;
	}

	dim3 blockSize(NUM_THREADS_PER_BLOCK);
	dim3 gridSize(numBlocks);
	

	/* Allocate surface and surfaceCopy in DEVICE */
	float *devSurface, *devSurfaceCopy, *dev_global_residual;

	hipMalloc((void **) &devSurface, numValues*sizeof(float));
	CHECK_ERROR("allocating devSurface");

	hipMalloc((void **) &devSurfaceCopy, numValues*sizeof(float));
	CHECK_ERROR("allocating devSurfaceCopy");

	hipMalloc((void **) &dev_global_residual, numValues*sizeof(float));
	CHECK_ERROR("allocating dev_global_residual");



    /* Auxiliar variables */
	int fRows = rows - 1;
	int fColumns = columns -1;
	float *temp;

	/* 3. Initialize surface */
	for( i=0; i<rows; i++ )
		for( j=0; j<paddingColumns; j++ )
			accessMat( surface, i, j ) = 0.0;



	/* 4. Simulation */
	int iter;
	int flag_stability = 0;
	int first_activation = 0;
	for( iter=0; iter<max_iter && ! flag_stability; iter++ ) {

		/* 4.1. Activate focal points */
		int num_deactivated = 0;
		for( i=0; i<num_focal; i++ ) {
			if ( focal[i].start == iter ) {
				focal[i].active = 1;
				if ( ! first_activation ) first_activation = 1;
			}
			// Count focal points already deactivated by a team
			if ( focal[i].active == 2 ) num_deactivated++;
		}

		if (!first_activation) continue;


		/* 4.2. Propagate heat (10 steps per each team movement) */
		float global_residual = 0.0f;
		int step;
		bool thresshold_passed = false;
		for( step=0; step<10; step++ )	{
			/* 4.2.1. Update heat on active focal points */
			for( i=0; i<num_focal; i++ ) {
				if ( focal[i].active != 1 ) continue;
				accessMat( surface, focal[i].x, focal[i].y ) = focal[i].heat;
			}

			/* 4.2.2. Copy values of the surface in ancillary structure (Skip borders) */
			temp = surface;
			surface = surfaceCopy;
			surfaceCopy = temp;

			/* Copy surface from HOST to DEVICE */
			hipMemcpy(devSurfaceCopy, surfaceCopy, realNumValues*sizeof(float), hipMemcpyHostToDevice);
			CHECK_ERROR("copying surfaceCopy from HOST to DEVICE");


			/* 4.2.3. Update surface values (skip borders) */
			propagate_kernel<<<gridSize, blockSize>>>(devSurface, devSurfaceCopy, rows, columns, paddingColumns);
			CHECK_ERROR("propagate_kernel");
			
			
			/* 4.2.4. Compute the maximum residual difference (absolute value) */
			if (num_deactivated == num_focal && !thresshold_passed) {
				
				/*Compute the residual difference of every position */
				difference_kernel<<<gridSize, blockSize>>>(dev_global_residual, devSurface, devSurfaceCopy, rows, columns, paddingColumns);
				CHECK_ERROR("difference_kernel");
	

				/* Reduction */
				int redSize = numValues;
				int sharedMemorySize = NUM_THREADS_PER_BLOCK * sizeof(float);
				
				while ( redSize > 1 ) 
				{
					int reductionBlocks = redSize / NUM_THREADS_PER_BLOCK;
					if (redSize % NUM_THREADS_PER_BLOCK != 0) {
						reductionBlocks++;
					}
					
					// Make the reduction of the residual difference in the corresponding level 
					reduce_max2_kernel<<< reductionBlocks, blockSize, sharedMemorySize >>>(dev_global_residual, redSize);
					CHECK_ERROR("reduction");

					// Update redSize to the number of blocks of the previous iteration 
					redSize = reductionBlocks;
				} 
				
				
				/* Copy the maximum residual difference from DEVICE to HOST */
				hipMemcpy(&global_residual, dev_global_residual, sizeof(float), hipMemcpyDeviceToHost);
				CHECK_ERROR("getting global_residual");

				if (global_residual >= THRESHOLD) {
					thresshold_passed = true;
				}				

			}// end IF 

			/* Copy surface from DEVICE to HOST */
			hipMemcpy(surface, devSurface,  realNumValues*sizeof(float), hipMemcpyDeviceToHost);
			CHECK_ERROR("copying surface from DEVICE to HOST");

		} // end 10 steps


		/* If the global residual is lower than THRESHOLD, we have reached enough stability, stop simulation at the end of this iteration */
		if( num_deactivated == num_focal && global_residual < THRESHOLD ) flag_stability = 1;

		if (num_focal != num_deactivated) {
			/* 4.3. Move teams */
			for( t=0; t<num_teams; t++ ) {
				/* 4.3.1. Choose nearest focal point */
				float distance = FLT_MAX;
				int target = -1;
				for( j=0; j<num_focal; j++ ) {
					if ( focal[j].active != 1 ) continue; // Skip non-active focal points
		
					float squared_local_distance = (focal[j].x - teams[t].x)*(focal[j].x - teams[t].x) + (focal[j].y - teams[t].y)*(focal[j].y - teams[t].y);
					if ( squared_local_distance < distance ) {
						distance = squared_local_distance;
						target = j;
					}
				}
				/* 4.3.2. Annotate target for the next stage */
				teams[t].target = target;

				/* 4.3.3. No active focal point to choose, no movement */
				if ( target == -1 ) continue; 

				/* 4.3.4. Move in the focal point direction */
				if ( teams[t].type == 1 ) { 
					// Type 1: Can move in diagonal
					if ( focal[target].x < teams[t].x ) teams[t].x--;
					else if ( focal[target].x > teams[t].x ) teams[t].x++;
					if ( focal[target].y < teams[t].y ) teams[t].y--;
					else if ( focal[target].y > teams[t].y ) teams[t].y++;
				}
				else if ( teams[t].type == 2 ) { 
					// Type 2: First in horizontal direction, then in vertical direction
					if ( focal[target].y < teams[t].y ) teams[t].y--;
					else if ( focal[target].y > teams[t].y ) teams[t].y++;
					else if ( focal[target].x < teams[t].x ) teams[t].x--;
					else if ( focal[target].x > teams[t].x ) teams[t].x++;
				}
				else {
					// Type 3: First in vertical direction, then in horizontal direction
					if ( focal[target].x < teams[t].x ) teams[t].x--;
					else if ( focal[target].x > teams[t].x ) teams[t].x++;
					else if ( focal[target].y < teams[t].y ) teams[t].y--;
					else if ( focal[target].y > teams[t].y ) teams[t].y++;
				}
			} // end team movements
		} // end IF (num_focals != num_deactivated)

		/* 4.4. Team actions */
		for( t=0; t<num_teams; t++ ) {
			/* 4.4.1. Deactivate the target focal point when it is reached */
			int target = teams[t].target;
			if ( target != -1 && focal[target].x == teams[t].x && focal[target].y == teams[t].y 
				&& focal[target].active == 1 )
				focal[target].active = 2;

			/* 4.4.2. Reduce heat in a circle around the team */
			
			// Influence area of fixed radius depending on type
			if ( teams[t].type == 1 ) {
				for( i=teams[t].x-RADIUS_TYPE_1; i<=teams[t].x+RADIUS_TYPE_1; i++ ) {
					for( j=teams[t].y-RADIUS_TYPE_1; j<=teams[t].y+RADIUS_TYPE_1; j++ ) {
						if ( i<1 || i>=fRows || j<1 || j>=fColumns ) continue; // Out of the heated surface
					
						float squared_distance = (teams[t].x - i)*(teams[t].x - i) + (teams[t].y - j)*(teams[t].y - j);
						if ( squared_distance <= SQUARED_RADIUS_TYPE_1 ) {
							accessMat( surface, i, j ) = accessMat( surface, i, j ) * 0.75; // Team efficiency factor
						}
					}
				}

			} else {
				for( i=teams[t].x-RADIUS_TYPE_2_3; i<=teams[t].x+RADIUS_TYPE_2_3; i++ ) {
					for( j=teams[t].y-RADIUS_TYPE_2_3; j<=teams[t].y+RADIUS_TYPE_2_3; j++ ) {
						if ( i<1 || i>=fRows || j<1 || j>=fColumns ) continue; // Out of the heated surface
					
						float squared_distance = (teams[t].x - i)*(teams[t].x - i) + (teams[t].y - j)*(teams[t].y - j);
						if ( squared_distance <= SQUARED_RADIUS_TYPE_2_3 ) {
							accessMat( surface, i, j ) = accessMat( surface, i, j ) * 0.75; // Team efficiency factor
						}
					}
				}
			}

			

			//team_actions_kernel<<<gridSize, blockSize>>>(devSurface, rows, columns, paddingColumns, teams[t].x, teams[t].y, squared_radius);
			//CHECK_ERROR("team actions");

		} // end team actions 

		/* Copy surface from DEVICE to host */
		//hipMemcpy(surface, devSurface, realNumValues*sizeof(float), hipMemcpyDeviceToHost);
		//CHECK_ERROR("copying surface from DEVICE to host 2");

#ifdef DEBUG
		/* 4.5. DEBUG: Print the current state of the simulation at the end of each iteration */
		print_status( iter, rows, columns, surface, num_teams, teams, num_focal, focal, global_residual );
#endif // DEBUG
	}
	
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	//printf("\n");
	/* 6.1. Total computation time */
	printf("\nTime: %lf\n", ttotal );
	/* 6.2. Results: Number of iterations, position of teams, residual heat on the focal points */
	printf("Result: %d", iter);
	/*
	for (i=0; i<num_teams; i++)
		printf(" %d %d", teams[i].x, teams[i].y );
	*/
	for (i=0; i<num_focal; i++)
		printf(" %.6f", accessMat( surface, focal[i].x, focal[i].y ) );
	printf("\n");

	/* 7. Free resources */	
	free( teams );
	free( focal );
	free( surface );
	free( surfaceCopy );

	/* 8. End */
	return 0;
}
